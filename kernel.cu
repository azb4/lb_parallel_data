#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

#include <stdio.h>

#define BLOCK_SIZE 32 
#define N 32

float getfloatnumber() {

	return (float)rand() / RAND_MAX;

}
__global__ void matMult(float* a, float* b, int n, float* c)

{

	int ROW = blockIdx.y * blockDim.y + threadIdx.y;

	int COL = blockIdx.x * blockDim.x + threadIdx.x;

	float tmpSum = 0;

	if (ROW < n && COL < n) {
		for (int i = 0; i < n; i++) {
			tmpSum += a[ROW * n + i] * b[i * n + COL];
		}
		c[ROW * n + COL] = tmpSum;
	}
	
}

int main(int argc, char* argv[])
{
	int numBytes = N * N * sizeof(float);

	// выделение памяти на хосте

	float* a = new float[N * N];
	float* b = new float[N * N];
	float* c = new float[N * N];

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			int k = N * i + j;
			a[k] = (float)k;
			b[k] = (float)k;
			c[k] = (float)k;
		}
	/*for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
		{
			int k = N * i + j;
			std::cout << a[k] << " ";
		}
		std::cout << "\n";
	}
	std::cout << "====================\n";
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
		{
			int k = N * i + j;
			std::cout << b[k] << " ";
		}
		std::cout << "\n";
	}*/

	// выделение памяти на девайсе

	float* adev = NULL;
	float* bdev = NULL;
	float* cdev = NULL;

	hipMalloc((void**)&adev, numBytes);
	hipMalloc((void**)&bdev, numBytes);
	hipMalloc((void**)&cdev, numBytes);


	//dim3 blocks;
	//dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	//if (N < threads.x || N < threads.y) {
	//	dim3 blocks(1, 1);
	//}
	//else {
	//	dim3 blocks(float(N) / float(threads.x), float(N / threads.y));
	//}


	hipEvent_t start, stop;
	float gpuTime = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);



	hipEventRecord(start, 0);
	hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);

	/*matMult <<< blocks, threads >>> (adev, bdev, N, cdev);*/

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocksPerGrid(1, 1);

	if (BLOCK_SIZE * BLOCK_SIZE > 1024) {

		threadsPerBlock.x = 1024;
		threadsPerBlock.y = 1024;

		blocksPerGrid.x = ceil(float(N) / float(threadsPerBlock.x));
		blocksPerGrid.y = ceil(float(N) / float(threadsPerBlock.y));

	}

	matMult << < blocksPerGrid, threadsPerBlock >> > (adev, bdev, N, cdev);

	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);

	/*std::cout << "====================\n";

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
		{
			int k = N * i + j;
			std::cout << c[k] << " ";
		}
		std::cout << "\n";
	}*/
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	std::cout<< c[N*N-1] << "\n";

	printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(adev);
	hipFree(bdev);
	hipFree(cdev);

	delete a;
	delete b;
	delete c;

	return 0;

}